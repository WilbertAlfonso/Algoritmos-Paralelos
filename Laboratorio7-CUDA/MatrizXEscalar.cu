#include "hip/hip_runtime.h"
#include <stdio.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <Windows.h>
#include <iostream>
using namespace std;
#define BLOCK 16
__global__
void matrizXescalarK(float* A, float B, float* C, int f,int c){
	int fila = blockIdx.y*blockDim.y + threadIdx.y;
	int columna = blockIdx.x*blockDim.x + threadIdx.x;
	if ((fila < f) && (columna < c)) {
		C[fila*c + columna] = B * A[fila*c + columna];
	}
}
void matrizXescalar(float* A, float B, float* C, int f,int c) {
	int size = f * c * sizeof(float);
	float *d_A, *d_C;
	hipMalloc((void **)&d_A, size);
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMalloc((void **)&d_C, size);
	int NumBlocks1 = c / BLOCK;
	int NumBlocks2 = f / BLOCK;
	if (c % BLOCK) NumBlocks1++;
	if (f%BLOCK) NumBlocks2++;
	dim3 dime1(NumBlocks1, NumBlocks2);
	dim3 dime2(BLOCK, BLOCK);
	matrizXescalarK << < dime1, dime2 >> > (d_A, B, d_C, f, c);
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	hipFree(d_A); hipFree(d_C);
}
void mostrarM(float* matriz, float fila, float columna)
{
	for (int x = 0; x < fila; x++)
	{
		for (int y = 0; y < columna; y++)
		{
			int puesto = x*columna + y;
			printf("%3.0f ", matriz[puesto]);
		}
		printf("\n");
	}
}
int main() {
	int fila = 17;
	int columna = 17;
	int espacio3 = 7;
	float* A = (float*)malloc(fila*columna*sizeof(float));
	float B = 2;
	float* C = (float*)malloc(fila*columna*sizeof(float));
	for (int i = 0; i < fila*columna; i++)
	{
		A[i] = i;
	}
	mostrarM(A, fila, columna);
	matrizXescalar(A, B, C, fila,columna);
	mostrarM(C, fila, columna);
	system("PAUSE");
	exit(0);
}
