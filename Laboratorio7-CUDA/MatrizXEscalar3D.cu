#include "hip/hip_runtime.h"
#include <stdio.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <Windows.h>
#include <iostream>
using namespace std;
#define BLOCK 10
__global__
void matrizXescalarK(float* A, float B, float* C, int f,int c,int e){
	int fila = blockIdx.y*blockDim.y + threadIdx.y;
	int columna = blockIdx.x*blockDim.x + threadIdx.x;
	int espacio = blockIdx.z*blockDim.z + threadIdx.z;
	if ((fila < f) && (columna < c) && (espacio < e)) {
		C[fila*c + columna + espacio*c*f] = B * A[fila*c + columna+espacio*c*f];
	}
}
void matrizXescalar(float* A, float B, float* C, int f,int c,int e) {
	int size = f * c * e *  sizeof(float);
	float *d_A, *d_C;
	hipMalloc((void **)&d_A, size);
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMalloc((void **)&d_C, size);
	int NumBlocks1 = c / BLOCK;
	int NumBlocks2 = f / BLOCK;
	int NumBlocks3 = e / BLOCK;
	if (c % BLOCK) NumBlocks1++;
	if (f % BLOCK) NumBlocks2++;
	if (e % BLOCK) NumBlocks3++;
	dim3 dime1(NumBlocks1, NumBlocks2, NumBlocks3);
	dim3 dime2(BLOCK, BLOCK, BLOCK);
	matrizXescalarK << < dime1, dime2 >> > (d_A, B, d_C, f, c,e);
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	hipFree(d_A); hipFree(d_C);
}
void mostrarM(float* matriz, float fila, float columna)
{
	for (int x = 0; x < fila; x++)
	{
		for (int y = 0; y < columna; y++)
		{
			int puesto = x*columna + y;
			printf("%3.0f ", matriz[puesto]);
		}
		printf("\n");
	}
}
int main() {
	int fila = 10;
	int columna = 10;
	int espacio3 = 10;
	float* A = (float*)malloc(fila*columna*espacio3*sizeof(float));
	float B = 2;
	float* C = (float*)malloc(fila*columna*espacio3*sizeof(float));
	for (int i = 0; i < fila*columna*espacio3; i++)
	{
		A[i] = i;
	}
	mostrarM(A, fila, columna*espacio3);
	matrizXescalar(A, B, C, fila,columna,espacio3);
	mostrarM(C, fila, columna*espacio3);
	system("PAUSE");
	exit(0);
}
