#include "hip/hip_runtime.h"
#include <stdio.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <Windows.h>
#include <iostream>

using namespace std;
__global__
void vecMultKernel(float* A, float* B, float* C, int n){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i<n){
		C[i] = 0;
		for (int j = 0; j<n; j++)
			C[i] += A[j*n + i] * B[j];
	}
}
void matrizXescalar(float* A, float* B, float* C, int n) {
	int size = n * n * sizeof(float);
	int sizevect = n * sizeof(float);
	float *d_A, *d_B, *d_C;
	hipMalloc((void **)&d_A, size);
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMalloc((void **)&d_B, sizevect);
	hipMemcpy(d_B, B, sizevect, hipMemcpyHostToDevice);
	hipMalloc((void **)&d_C, sizevect);
	vecMultKernel <<< ceil((n*n) / 256.0), 256 >>> (d_A, d_B, d_C, n);
	hipMemcpy(C, d_C, sizevect, hipMemcpyDeviceToHost);
	hipFree(d_A); hipFree(d_B); hipFree(d_C);
}
void mostrarM(float* matriz, float fila, float columna)
{
	for (int x = 0; x < fila; x++)
	{
		for (int y = 0; y < columna; y++)
		{
			int puesto = x*columna + y;
			printf("%3.0f ", matriz[puesto]);
		}
		printf("\n");
	}
}
int main() {
	int fila = 3;
	int columna = 3;
	float* A = (float*)malloc(fila*columna*sizeof(float));
	float* B = (float*)malloc(fila*sizeof(float));
	float* C = (float*)malloc(fila*sizeof(float));
	for (int i = 0; i < fila*columna; i++)
	{
		A[i] = i;
	}
	for (int i = 0; i < fila; i++)
	{
		B[i] = i;
	}
	mostrarM(A, fila, columna);
	cout << endl;
	mostrarM(B, 1, columna);
	matrizXescalar(A, B, C, fila);
	mostrarM(C, 1, columna);
	system("PAUSE");
	exit(0);
}
