#include "hip/hip_runtime.h"
#include <stdio.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <Windows.h>
#include <iostream>
using namespace std;
__global__
void matrixAddKernel(float* A, float* B, float* C, int n){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < n*n) C[i] = A[i] + B[i];
}
__global__
void matrixAddKernel2(float* A, float* B, float* C, int n){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < n){
		for (int j = i * n; j < i * n + n; j++)
			C[j] = A[j] + B[j];
	}
}
__global__
void matrixAddKernel3(float* A, float* B, float* C, int n){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i<n)
	for (int j = i; j < n*n; j += n)
		C[j] = A[j] + B[j];
}
void matrixAdd(float* A, float* B, float* C, int n) {
	int size = n * n * sizeof(float);
	float *d_A, *d_B, *d_C;
	hipMalloc((void **)&d_A, size);
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMalloc((void **)&d_B, size);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
	hipMalloc((void **)&d_C, size);
	matrixAddKernel3 <<< ceil((n*n) / 256.0), 256 >>> (d_A, d_B, d_C, n);
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	hipFree(d_A); hipFree(d_B); hipFree(d_C);
}
void mostrarM(float* matriz, float fila, float columna)
{
	for (int x = 0; x < fila; x++)
	{
		for (int y = 0; y < columna; y++)
		{
			int puesto = x*columna + y;
			printf("%3.0f ", matriz[puesto]);
		}
		printf("\n");
	}
}
int main() {
	int fila = 5;
	float* A = (float*)malloc(fila*fila*sizeof(float));
	float* B = (float*)malloc(fila*fila*sizeof(float));
	float* C = (float*)malloc(fila*fila*sizeof(float));
	for (int i = 0; i < fila*fila; i++)
	{
		A[i] = i;
		B[i] = i;
	}
	matrixAdd(A, B, C, fila);
	mostrarM(C, fila, fila);
	system("PAUSE");
	exit(0);
}
